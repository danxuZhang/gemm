#include "gemm.h"
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

namespace gemm {

void handleCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", message, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void handleCublasError(hipblasStatus_t status, const char* message) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS error: %s\n", message);
        exit(EXIT_FAILURE);
    }
}

void dgemm_cublas(int m, int n, int k, double alpha, const double* a, int lda,
                const double* b, int ldb, double beta, double* c, int ldc) {
    // Create cuBLAS handle
    hipblasHandle_t handle;
    handleCublasError(hipblasCreate(&handle), "hipblasCreate failed");

    hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_NOT_ALLOWED);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    // Allocate device memory
    double *d_a, *d_b, *d_c;
    handleCudaError(hipMalloc(&d_a, m * k * sizeof(double)), "hipMalloc d_a failed");
    handleCudaError(hipMalloc(&d_b, k * n * sizeof(double)), "hipMalloc d_b failed");
    handleCudaError(hipMalloc(&d_c, m * n * sizeof(double)), "hipMalloc d_c failed");

    // Copy data to device
    handleCudaError(hipMemcpy(d_a, a, m * k * sizeof(double), hipMemcpyHostToDevice),
                   "hipMemcpy a failed");
    handleCudaError(hipMemcpy(d_b, b, k * n * sizeof(double), hipMemcpyHostToDevice),
                   "hipMemcpy b failed");

    // Perform DGEMM
    // Note: cuBLAS uses column-major order, while our input is row-major
    // So we compute B * A instead of A * B and adjust the parameters accordingly
    handleCublasError(
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    n, m, k,
                    &alpha,
                    d_b, ldb,
                    d_a, lda,
                    &beta,
                    d_c, ldc),
        "hipblasDgemm failed");

    // Copy result back to host
    handleCudaError(hipMemcpy(c, d_c, m * n * sizeof(double), hipMemcpyDeviceToHost),
                   "hipMemcpy c failed");

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);
}

} // namespace gemm